#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "backward.h"
#include "auxiliary.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

# define M_PI 3.14159265358979323846

// Backward pass for conversion of spherical harmonics to RGB for
// each Gaussian.
//! We dont need it.
__device__ void computeColorFromSH(int idx, int deg, int max_coeffs, const glm::vec3* means, glm::vec3 campos, const float* shs, const bool* clamped, const glm::vec3* dL_dcolor, glm::vec3* dL_dmeans, glm::vec3* dL_dshs)
{
	// Compute intermediate values, as it is done during forward
	glm::vec3 pos = means[idx];
	glm::vec3 dir_orig = pos - campos;
	glm::vec3 dir = dir_orig / glm::length(dir_orig);

	glm::vec3* sh = ((glm::vec3*)shs) + idx * max_coeffs;

	// Use PyTorch rule for clamping: if clamping was applied,
	// gradient becomes 0.
	glm::vec3 dL_dRGB = dL_dcolor[idx];
	dL_dRGB.x *= clamped[3 * idx + 0] ? 0 : 1;
	dL_dRGB.y *= clamped[3 * idx + 1] ? 0 : 1;
	dL_dRGB.z *= clamped[3 * idx + 2] ? 0 : 1;

	glm::vec3 dRGBdx(0, 0, 0);
	glm::vec3 dRGBdy(0, 0, 0);
	glm::vec3 dRGBdz(0, 0, 0);
	float x = dir.x;
	float y = dir.y;
	float z = dir.z;

	// Target location for this Gaussian to write SH gradients to
	glm::vec3* dL_dsh = dL_dshs + idx * max_coeffs;

	// No tricks here, just high school-level calculus.
	float dRGBdsh0 = SH_C0;
	dL_dsh[0] = dRGBdsh0 * dL_dRGB;
	if (deg > 0)
	{
		float dRGBdsh1 = -SH_C1 * y;
		float dRGBdsh2 = SH_C1 * z;
		float dRGBdsh3 = -SH_C1 * x;
		dL_dsh[1] = dRGBdsh1 * dL_dRGB;
		dL_dsh[2] = dRGBdsh2 * dL_dRGB;
		dL_dsh[3] = dRGBdsh3 * dL_dRGB;

		dRGBdx = -SH_C1 * sh[3];
		dRGBdy = -SH_C1 * sh[1];
		dRGBdz = SH_C1 * sh[2];

		if (deg > 1)
		{
			float xx = x * x, yy = y * y, zz = z * z;
			float xy = x * y, yz = y * z, xz = x * z;

			float dRGBdsh4 = SH_C2[0] * xy;
			float dRGBdsh5 = SH_C2[1] * yz;
			float dRGBdsh6 = SH_C2[2] * (2.f * zz - xx - yy);
			float dRGBdsh7 = SH_C2[3] * xz;
			float dRGBdsh8 = SH_C2[4] * (xx - yy);
			dL_dsh[4] = dRGBdsh4 * dL_dRGB;
			dL_dsh[5] = dRGBdsh5 * dL_dRGB;
			dL_dsh[6] = dRGBdsh6 * dL_dRGB;
			dL_dsh[7] = dRGBdsh7 * dL_dRGB;
			dL_dsh[8] = dRGBdsh8 * dL_dRGB;

			dRGBdx += SH_C2[0] * y * sh[4] + SH_C2[2] * 2.f * -x * sh[6] + SH_C2[3] * z * sh[7] + SH_C2[4] * 2.f * x * sh[8];
			dRGBdy += SH_C2[0] * x * sh[4] + SH_C2[1] * z * sh[5] + SH_C2[2] * 2.f * -y * sh[6] + SH_C2[4] * 2.f * -y * sh[8];
			dRGBdz += SH_C2[1] * y * sh[5] + SH_C2[2] * 2.f * 2.f * z * sh[6] + SH_C2[3] * x * sh[7];

			if (deg > 2)
			{
				float dRGBdsh9 = SH_C3[0] * y * (3.f * xx - yy);
				float dRGBdsh10 = SH_C3[1] * xy * z;
				float dRGBdsh11 = SH_C3[2] * y * (4.f * zz - xx - yy);
				float dRGBdsh12 = SH_C3[3] * z * (2.f * zz - 3.f * xx - 3.f * yy);
				float dRGBdsh13 = SH_C3[4] * x * (4.f * zz - xx - yy);
				float dRGBdsh14 = SH_C3[5] * z * (xx - yy);
				float dRGBdsh15 = SH_C3[6] * x * (xx - 3.f * yy);
				dL_dsh[9] = dRGBdsh9 * dL_dRGB;
				dL_dsh[10] = dRGBdsh10 * dL_dRGB;
				dL_dsh[11] = dRGBdsh11 * dL_dRGB;
				dL_dsh[12] = dRGBdsh12 * dL_dRGB;
				dL_dsh[13] = dRGBdsh13 * dL_dRGB;
				dL_dsh[14] = dRGBdsh14 * dL_dRGB;
				dL_dsh[15] = dRGBdsh15 * dL_dRGB;

				dRGBdx += (
					SH_C3[0] * sh[9] * 3.f * 2.f * xy +
					SH_C3[1] * sh[10] * yz +
					SH_C3[2] * sh[11] * -2.f * xy +
					SH_C3[3] * sh[12] * -3.f * 2.f * xz +
					SH_C3[4] * sh[13] * (-3.f * xx + 4.f * zz - yy) +
					SH_C3[5] * sh[14] * 2.f * xz +
					SH_C3[6] * sh[15] * 3.f * (xx - yy));

				dRGBdy += (
					SH_C3[0] * sh[9] * 3.f * (xx - yy) +
					SH_C3[1] * sh[10] * xz +
					SH_C3[2] * sh[11] * (-3.f * yy + 4.f * zz - xx) +
					SH_C3[3] * sh[12] * -3.f * 2.f * yz +
					SH_C3[4] * sh[13] * -2.f * xy +
					SH_C3[5] * sh[14] * -2.f * yz +
					SH_C3[6] * sh[15] * -3.f * 2.f * xy);

				dRGBdz += (
					SH_C3[1] * sh[10] * xy +
					SH_C3[2] * sh[11] * 4.f * 2.f * yz +
					SH_C3[3] * sh[12] * 3.f * (2.f * zz - xx - yy) +
					SH_C3[4] * sh[13] * 4.f * 2.f * xz +
					SH_C3[5] * sh[14] * (xx - yy));
			}
		}
	}

	// The view direction is an input to the computation. View direction
	// is influenced by the Gaussian's mean, so SHs gradients
	// must propagate back into 3D position.
	glm::vec3 dL_ddir(glm::dot(dRGBdx, dL_dRGB), glm::dot(dRGBdy, dL_dRGB), glm::dot(dRGBdz, dL_dRGB));

	// Account for normalization of direction
	float3 dL_dmean = dnormvdv(float3{ dir_orig.x, dir_orig.y, dir_orig.z }, float3{ dL_ddir.x, dL_ddir.y, dL_ddir.z });

	// Gradients of loss w.r.t. Gaussian means, but only the portion 
	// that is caused because the mean affects the view-dependent color.
	// Additional mean gradient is accumulated in below methods.
	dL_dmeans[idx] += glm::vec3(dL_dmean.x, dL_dmean.y, dL_dmean.z);
}

// Backward version of INVERSE 2D covariance matrix computation
// (due to length launched as separate kernel before other 
// backward steps contained in preprocess)
__global__ void computeCov2DCUDA(int P,
	const float3* means,
	const int* radii,
	const float* cov3Ds,
	const float h_x, float h_y,
	const float tan_fovx, float tan_fovy,
	const float* view_matrix,
	const float* dL_dconics,
	const float* dL_dmus,
	float3* dL_dmeans,
	float* dL_dcov,
	const int mode)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P || !(radii[idx] > 0))
		return;

	// Reading location of 3D covariance for this Gaussian
	const float* cov3D = cov3Ds + 6 * idx;

	// Fetch gradients, recompute 2D covariance and relevant 
	// intermediate forward results needed in the backward.
	float3 mean = means[idx];
	float3 dL_dconic = { dL_dconics[4 * idx], dL_dconics[4 * idx + 1], dL_dconics[4 * idx + 3] };
	float dL_dmu = dL_dmus[idx]; 
	float3 t = transformPoint4x3(mean, view_matrix);

	glm::mat3 J;
	float x_grad_mul, y_grad_mul;
	if (mode == 0){  //! parallel beam
		const float limx = 1.3f;
		const float limy = 1.3f;
		t.x = min(limx, max(-limx, t.x));
		t.y = min(limx, max(-limx, t.y));
		
		x_grad_mul = t.x < -limx || t.x > limx ? 0 : 1;
		y_grad_mul = t.y < -limy || t.y > limy ? 0 : 1;

		J = glm::mat3(
		h_x, 0.0f, 0.0f,
		0.0f, h_y, 0.0f,
		0, 0, 1.0f);
	}
	else  //! cone beam
	{
		const float limx = 1.3f * tan_fovx;
		const float limy = 1.3f * tan_fovy;
		const float txtz = t.x / t.z;
		const float tytz = t.y / t.z;
		t.x = min(limx, max(-limx, txtz)) * t.z;
		t.y = min(limy, max(-limy, tytz)) * t.z;
		
		x_grad_mul = txtz < -limx || txtz > limx ? 0 : 1;
		y_grad_mul = tytz < -limy || tytz > limy ? 0 : 1;

		const float l = sqrt(t.x * t.x +  t.y * t.y + t.z * t.z);
		J = glm::mat3(
			h_x / t.z, 0.0f, -(h_x * t.x) / (t.z * t.z),
			0.0f, h_y / t.z, -(h_y * t.y) / (t.z * t.z),
			t.x / l, t.y / l, t.z / l);  //! We need last row for computation.
	}

	glm::mat3 W = glm::mat3(
		view_matrix[0], view_matrix[4], view_matrix[8],
		view_matrix[1], view_matrix[5], view_matrix[9],
		view_matrix[2], view_matrix[6], view_matrix[10]);

	glm::mat3 Vrk = glm::mat3(
		cov3D[0], cov3D[1], cov3D[2],
		cov3D[1], cov3D[3], cov3D[4],
		cov3D[2], cov3D[4], cov3D[5]);

	glm::mat3 M = W * J;

	glm::mat3 cov = glm::transpose(M) * glm::transpose(Vrk) * M;

	// Use helper variables for 2D covariance entries. More compact.
	float hata = cov[0][0] += 0.0f;
	float hatb = cov[0][1];
	float hatc = cov[0][2];
	float hatd = cov[1][1] += 0.0f;
	float hate = cov[1][2];
	float hatf = cov[2][2];

	float dL_dhata = 0, dL_dhatb = 0, dL_dhatc = 0, dL_dhatd = 0, dL_dhate = 0, dL_dhatf = 0;
	float denom = hata * hatd - hatb * hatb;
	float denom2inv = 1.0f / ((denom * denom) + 0.0000001f);
	float diamond = hata * hatd - hatb * hatb;

	//! mu part gradient
	float circ = hata * hatd * hatf + 2 * hatb * hatc * hate - hata * hate * hate - hatf * hatb * hatb - hatd * hatc * hatc;
	float mu_square = 2 * M_PI * circ / diamond;
	float mu = 0.0f;
	if (mu_square > 0.0f){
		mu = sqrt(2 * M_PI * circ / diamond);
	}
	float pi_mu= M_PI / (mu + 0.0000001f);
	float circ_diamond = circ / diamond;

	if (denom2inv != 0.0f && mu != 0.0f)
	{
		// exp(*) part gradient
		dL_dhata = denom2inv * (-hatd * hatd * dL_dconic.x + hatb * hatd * dL_dconic.y + (denom - hata * hatd) * dL_dconic.z);  // We remove 2 here because in render we do not *0.5
		dL_dhatd = denom2inv * (-hata * hata * dL_dconic.z + hata * hatb * dL_dconic.y + (denom - hata * hatd) * dL_dconic.x);  // We remove 2 here because in render we do not *0.5
		dL_dhatb = denom2inv * (2 * hatb * hatd * dL_dconic.x - (denom + 2 * hatb * hatb) * dL_dconic.y + 2 * hata * hatb * dL_dconic.z);

		dL_dhata += pi_mu * ((hatd * hatf - hate * hate) / diamond -  hatd * circ_diamond / diamond) * dL_dmu;
		dL_dhatb += pi_mu * ((2 * hatc * hate - 2 * hatf * hatb) / diamond + 2 * hatb * circ_diamond / diamond) * dL_dmu;
		dL_dhatc += pi_mu * ((2 * hatb * hate - 2 * hatd * hatc) / diamond) * dL_dmu;
		dL_dhatd += pi_mu * ((hata * hatf - hatc * hatc) / diamond -  hata *circ_diamond / diamond) * dL_dmu;
		dL_dhate += pi_mu * ((2 * hatb * hatc - 2 * hata * hate) / diamond) * dL_dmu;
		dL_dhatf += pi_mu * ((hata * hatd - hatb * hatb) / diamond) * dL_dmu;

		// Gradients of loss L w.r.t. each 3D covariance matrix (Vrk) entry
		// dL_da
		dL_dcov[6 * idx + 0] += M[0][0]*M[0][0]*dL_dhata + M[0][0]*M[1][0]*dL_dhatb + M[0][0]*M[2][0]*dL_dhatc + M[1][0]*M[1][0]*dL_dhatd + M[1][0]*M[2][0]*dL_dhate + M[2][0]*M[2][0]*dL_dhatf;
		// dL_dd
		dL_dcov[6 * idx + 3] += M[0][1]*M[0][1]*dL_dhata + M[0][1]*M[1][1]*dL_dhatb + M[0][1]*M[2][1]*dL_dhatc + M[1][1]*M[1][1]*dL_dhatd + M[1][1]*M[2][1]*dL_dhate + M[2][1]*M[2][1]*dL_dhatf;
		// dL_df
		dL_dcov[6 * idx + 5] += M[0][2]*M[0][2]*dL_dhata + M[0][2]*M[1][2]*dL_dhatb + M[0][2]*M[2][2]*dL_dhatc + M[1][2]*M[1][2]*dL_dhatd + M[1][2]*M[2][2]*dL_dhate + M[2][2]*M[2][2]*dL_dhatf;
		
		// dL_db
		dL_dcov[6 * idx + 1] += 2*M[0][0]*M[0][1]*dL_dhata + (M[0][1]*M[1][0]+M[0][0]*M[1][1])*dL_dhatb + (M[0][1]*M[2][0]+M[0][0]*M[2][1])*dL_dhatc + 2*M[1][0]*M[1][1]*dL_dhatd + (M[1][1]*M[2][0]+M[1][0]*M[2][1])*dL_dhate + 2*M[2][0]*M[2][1]*dL_dhatf;
		// dL_dc
		dL_dcov[6 * idx + 2] += 2*M[0][0]*M[0][2]*dL_dhata + (M[0][2]*M[1][0]+M[0][0]*M[1][2])*dL_dhatb + (M[0][2]*M[2][0]+M[0][0]*M[2][2])*dL_dhatc + 2*M[1][0]*M[1][2]*dL_dhatd + (M[1][2]*M[2][0]+M[1][0]*M[2][2])*dL_dhate + 2*M[2][0]*M[2][2]*dL_dhatf;
		// dL_de
		dL_dcov[6 * idx + 4] += 2*M[0][1]*M[0][2]*dL_dhata + (M[0][2]*M[1][1]+M[0][1]*M[1][2])*dL_dhatb + (M[0][2]*M[2][1]+M[0][1]*M[2][2])*dL_dhatc + 2*M[1][1]*M[1][2]*dL_dhatd + (M[1][2]*M[2][1]+M[1][1]*M[2][2])*dL_dhate + 2*M[2][1]*M[2][2]*dL_dhatf;
	}
	else
	{
		for (int i = 0; i < 6; i++)
			dL_dcov[6 * idx + i] = 0;
	}

	if (mode == 1){
		// Gradients of loss w.r.t. M
		// cov2D = transpose(M) * transpose(Vrk) * M;

		float a = cov3D[0];
		float b = cov3D[1];
		float c = cov3D[2];
		float d = cov3D[3];
		float e = cov3D[4];
		float f = cov3D[5];

		float dL_dM00 = 2*(M[0][0]*a+M[0][1]*b + M[0][2]*c)*dL_dhata + (M[1][0]*a+M[1][1]*b+M[1][2]*c)*dL_dhatb + (M[2][0]*a+M[2][1]*b+M[2][2]*c)*dL_dhatc;
		float dL_dM01 = 2*(M[0][0]*b+M[0][1]*d + M[0][2]*e)*dL_dhata + (M[1][0]*b+M[1][1]*d+M[1][2]*e)*dL_dhatb + (M[2][0]*b+M[2][1]*d+M[2][2]*e)*dL_dhatc;
		float dL_dM02 = 2*(M[0][0]*c+M[0][1]*e + M[0][2]*f)*dL_dhata + (M[1][0]*c+M[1][1]*e+M[1][2]*f)*dL_dhatb + (M[2][0]*c+M[2][1]*e+M[2][2]*f)*dL_dhatc;

		float dL_dM10 = (M[0][0]*a+M[0][1]*b+M[0][2]*c)*dL_dhatb + 2*(M[1][0]*a+M[1][1]*b+M[1][2]*c)*dL_dhatd + (M[2][0]*a+M[2][1]*b+M[2][2]*c)*dL_dhate;
		float dL_dM11 = (M[0][0]*b+M[0][1]*d+M[0][2]*e)*dL_dhatb + 2*(M[1][0]*b+M[1][1]*d+M[1][2]*e)*dL_dhatd + (M[2][0]*b+M[2][1]*d+M[2][2]*e)*dL_dhate;
		float dL_dM12 = (M[0][0]*c+M[0][1]*e+M[0][2]*f)*dL_dhatb + 2*(M[1][0]*c+M[1][1]*e+M[1][2]*f)*dL_dhatd + (M[2][0]*c+M[2][1]*e+M[2][2]*f)*dL_dhate;

		float dL_dM20 = (M[0][0]*a+M[0][1]*b+M[0][2]*c)*dL_dhatc + (M[1][0]*a+M[1][1]*b+M[1][2]*c)*dL_dhate + 2*(M[2][0]*a+M[2][1]*b+M[2][2]*c)*dL_dhatf;
		float dL_dM21 = (M[0][0]*b+M[0][1]*d+M[0][2]*e)*dL_dhatc + (M[1][0]*b+M[1][1]*d+M[1][2]*e)*dL_dhate + 2*(M[2][0]*b+M[2][1]*d+M[2][2]*e)*dL_dhatf;
		float dL_dM22 = (M[0][0]*c+M[0][1]*e+M[0][2]*f)*dL_dhatc + (M[1][0]*c+M[1][1]*e+M[1][2]*f)*dL_dhate + 2*(M[2][0]*c+M[2][1]*e+M[2][2]*f)*dL_dhatf;

		float dL_dJ00 = W[0][0]*dL_dM00 + W[0][1]*dL_dM01 + W[0][2]*dL_dM02;
		float dL_dJ02 = W[2][0]*dL_dM00 + W[2][1]*dL_dM01 + W[2][2]*dL_dM02;
		float dL_dJ11 = W[1][0]*dL_dM10 + W[1][1]*dL_dM11 + W[1][2]*dL_dM12;
		float dL_dJ12 = W[2][0]*dL_dM10 + W[2][1]*dL_dM11 + W[2][2]*dL_dM12;
		float dL_dJ20 = W[0][0]*dL_dM20 + W[0][1]*dL_dM21 + W[0][2]*dL_dM22;
		float dL_dJ21 = W[1][0]*dL_dM20 + W[1][1]*dL_dM21 + W[1][2]*dL_dM22;
		float dL_dJ22 = W[2][0]*dL_dM20 + W[2][1]*dL_dM21 + W[2][2]*dL_dM22;

		float tx = t.x;
		float ty = t.y;
		float tz = t.z;
		float inv_tz = 1.f / tz;
		float inv_tz2 = inv_tz * inv_tz;
		float inv_tz3 = inv_tz2 * inv_tz;
		float circledcirc = sqrt(tx * tx + ty * ty + tz * tz);
		float inv_circledcirc3 = 1 / (circledcirc * circledcirc * circledcirc);
		float dL_dtx = x_grad_mul * (-h_x*inv_tz2*dL_dJ02 + (1/circledcirc - tx*tx*inv_circledcirc3)*dL_dJ20 - tx*ty*inv_circledcirc3*dL_dJ21 - tx*tz*inv_circledcirc3*dL_dJ22);
		float dL_dty = y_grad_mul * (-h_y*inv_tz2*dL_dJ12 - tx*ty*inv_circledcirc3*dL_dJ20 + (1/circledcirc - ty*ty*inv_circledcirc3)*dL_dJ21 - ty*tz*inv_circledcirc3*dL_dJ22);
		float dL_dtz = -h_x*inv_tz2*dL_dJ00 + 2*h_x*tx*inv_tz3*dL_dJ02 - h_y*inv_tz2*dL_dJ11 + 2*h_y*ty*inv_tz3*dL_dJ12 - tx*tz*inv_circledcirc3*dL_dJ20 - ty*tz*inv_circledcirc3*dL_dJ21 + (1/circledcirc-tz*tz*inv_circledcirc3)*dL_dJ22;

		float3 dL_dmean = transformVec4x3Transpose({ dL_dtx, dL_dty, dL_dtz }, view_matrix);

		// Gradients of loss w.r.t. Gaussian means, but only the portion 
		// that is caused because the mean affects the covariance matrix.
		// Additional mean gradient is accumulated in BACKWARD::preprocess.
		dL_dmeans[idx] = dL_dmean;

	}
}

// Backward pass for the conversion of scale and rotation to a 
// 3D covariance matrix for each Gaussian. 
__device__ void computeCov3D(int idx, const glm::vec3 scale, float mod, const glm::vec4 rot, const float* dL_dcov3Ds, glm::vec3* dL_dscales, glm::vec4* dL_drots)
{
	// Recompute (intermediate) results for the 3D covariance computation.
	glm::vec4 q = rot;// / glm::length(rot);
	float r = q.x;
	float x = q.y;
	float y = q.z;
	float z = q.w;

	glm::mat3 R = glm::mat3(
		1.f - 2.f * (y * y + z * z), 2.f * (x * y - r * z), 2.f * (x * z + r * y),
		2.f * (x * y + r * z), 1.f - 2.f * (x * x + z * z), 2.f * (y * z - r * x),
		2.f * (x * z - r * y), 2.f * (y * z + r * x), 1.f - 2.f * (x * x + y * y)
	);

	glm::mat3 S = glm::mat3(1.0f);

	glm::vec3 s = mod * scale;
	S[0][0] = s.x;
	S[1][1] = s.y;
	S[2][2] = s.z;

	glm::mat3 M = S * R;

	const float* dL_dcov3D = dL_dcov3Ds + 6 * idx;

	glm::vec3 dunc(dL_dcov3D[0], dL_dcov3D[3], dL_dcov3D[5]);
	glm::vec3 ounc = 0.5f * glm::vec3(dL_dcov3D[1], dL_dcov3D[2], dL_dcov3D[4]);

	// Convert per-element covariance loss gradients to matrix form
	glm::mat3 dL_dSigma = glm::mat3(
		dL_dcov3D[0], 0.5f * dL_dcov3D[1], 0.5f * dL_dcov3D[2],
		0.5f * dL_dcov3D[1], dL_dcov3D[3], 0.5f * dL_dcov3D[4],
		0.5f * dL_dcov3D[2], 0.5f * dL_dcov3D[4], dL_dcov3D[5]
	);

	// Compute loss gradient w.r.t. matrix M
	// dSigma_dM = 2 * M
	glm::mat3 dL_dM = 2.0f * M * dL_dSigma;

	glm::mat3 Rt = glm::transpose(R);
	glm::mat3 dL_dMt = glm::transpose(dL_dM);

	// Gradients of loss w.r.t. scale
	glm::vec3* dL_dscale = dL_dscales + idx;
	dL_dscale->x = glm::dot(Rt[0], dL_dMt[0]);
	dL_dscale->y = glm::dot(Rt[1], dL_dMt[1]);
	dL_dscale->z = glm::dot(Rt[2], dL_dMt[2]);

	dL_dMt[0] *= s.x;
	dL_dMt[1] *= s.y;
	dL_dMt[2] *= s.z;

	// Gradients of loss w.r.t. normalized quaternion
	glm::vec4 dL_dq;
	dL_dq.x = 2 * z * (dL_dMt[0][1] - dL_dMt[1][0]) + 2 * y * (dL_dMt[2][0] - dL_dMt[0][2]) + 2 * x * (dL_dMt[1][2] - dL_dMt[2][1]);
	dL_dq.y = 2 * y * (dL_dMt[1][0] + dL_dMt[0][1]) + 2 * z * (dL_dMt[2][0] + dL_dMt[0][2]) + 2 * r * (dL_dMt[1][2] - dL_dMt[2][1]) - 4 * x * (dL_dMt[2][2] + dL_dMt[1][1]);
	dL_dq.z = 2 * x * (dL_dMt[1][0] + dL_dMt[0][1]) + 2 * r * (dL_dMt[2][0] - dL_dMt[0][2]) + 2 * z * (dL_dMt[1][2] + dL_dMt[2][1]) - 4 * y * (dL_dMt[2][2] + dL_dMt[0][0]);
	dL_dq.w = 2 * r * (dL_dMt[0][1] - dL_dMt[1][0]) + 2 * x * (dL_dMt[2][0] + dL_dMt[0][2]) + 2 * y * (dL_dMt[1][2] + dL_dMt[2][1]) - 4 * z * (dL_dMt[1][1] + dL_dMt[0][0]);

	// Gradients of loss w.r.t. unnormalized quaternion
	float4* dL_drot = (float4*)(dL_drots + idx);
	*dL_drot = float4{ dL_dq.x, dL_dq.y, dL_dq.z, dL_dq.w };//dnormvdv(float4{ rot.x, rot.y, rot.z, rot.w }, float4{ dL_dq.x, dL_dq.y, dL_dq.z, dL_dq.w });
}

// Backward pass of the preprocessing steps, except
// for the covariance computation and inversion
// (those are handled by a previous kernel call)
template<int C>
__global__ void preprocessCUDA(
	int P,
	const float3* means,
	const int* radii,
	const glm::vec3* scales,
	const glm::vec4* rotations,
	const float scale_modifier,
	const float* proj,
	const glm::vec3* campos,
	const float3* dL_dmean2D,
	glm::vec3* dL_dmeans,
	float* dL_dcov3D,
	glm::vec3* dL_dscale,
	glm::vec4* dL_drot)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P || !(radii[idx] > 0))
		return;

	float3 m = means[idx];

	// Taking care of gradients from the screenspace points
	float4 m_hom = transformPoint4x4(m, proj);
	float m_w = 1.0f / (m_hom.w + 0.0000001f);

	// Compute loss gradient w.r.t. 3D means due to gradients of 2D means
	// from rendering procedure
	glm::vec3 dL_dmean;
	float mul1 = (proj[0] * m.x + proj[4] * m.y + proj[8] * m.z + proj[12]) * m_w * m_w;
	float mul2 = (proj[1] * m.x + proj[5] * m.y + proj[9] * m.z + proj[13]) * m_w * m_w;
	dL_dmean.x = (proj[0] * m_w - proj[3] * mul1) * dL_dmean2D[idx].x + (proj[1] * m_w - proj[3] * mul2) * dL_dmean2D[idx].y;
	dL_dmean.y = (proj[4] * m_w - proj[7] * mul1) * dL_dmean2D[idx].x + (proj[5] * m_w - proj[7] * mul2) * dL_dmean2D[idx].y;
	dL_dmean.z = (proj[8] * m_w - proj[11] * mul1) * dL_dmean2D[idx].x + (proj[9] * m_w - proj[11] * mul2) * dL_dmean2D[idx].y;

	// That's the second part of the mean gradient. Previous computation
	// of cov2D and following SH conversion also affects it.
	dL_dmeans[idx] += dL_dmean;

	// Compute gradient updates due to computing covariance from scale/rotation
	if (scales)
		computeCov3D(idx, scales[idx], scale_modifier, rotations[idx], dL_dcov3D, dL_dscale, dL_drot);
}

// Backward version of the rendering procedure.
template <uint32_t C>
__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
renderCUDA(
	const uint2* __restrict__ ranges,
	const uint32_t* __restrict__ point_list,
	int W, int H,
	const float2* __restrict__ points_xy_image,
	const float4* __restrict__ conic_opacity,
	const float* __restrict__ mus,
	const uint32_t* __restrict__ n_contrib,
	const float* __restrict__ dL_dpixels,
	float3* __restrict__ dL_dmean2D,
	float4* __restrict__ dL_dconic2D,
	float* __restrict__ dL_dopacity,
	float* __restrict__ dL_dmu)
{	
	// We rasterize again. Compute necessary block info.
	auto block = cg::this_thread_block();
	const uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X;
	const uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	const uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };
	const uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	const uint32_t pix_id = W * pix.y + pix.x;
	float2 pixf = { (float)pix.x, (float)pix.y};

	const bool inside = pix.x < W&& pix.y < H;
	const uint2 range = ranges[block.group_index().y * horizontal_blocks + block.group_index().x];

	const int rounds = ((range.y - range.x + BLOCK_SIZE - 1) / BLOCK_SIZE);

	bool done = !inside;
	int toDo = range.y - range.x;

	__shared__ int collected_id[BLOCK_SIZE];
	__shared__ float2 collected_xy[BLOCK_SIZE];
	__shared__ float4 collected_conic_opacity[BLOCK_SIZE];
	__shared__ float collected_mu[BLOCK_SIZE];

	// We start from the back. The ID of the last contributing
	// Gaussian is known from each pixel from the forward.
	uint32_t contributor = toDo;
	const int last_contributor = inside ? n_contrib[pix_id] : 0;
    
	float dL_dpixel[C];
	if (inside)
		for (int i = 0; i < C; i++)
			dL_dpixel[i] = dL_dpixels[i * H * W + pix_id];

	// Gradient of pixel coordinate w.r.t. normalized 
	// screen-space viewport corrdinates (-1 to 1)
	const float ddelx_dx = 0.5 * W;
	const float ddely_dy = 0.5 * H;

	// Traverse all Gaussians
	for (int i = 0; i < rounds; i++, toDo -= BLOCK_SIZE)
	{
		// Load auxiliary data into shared memory, start in the BACK
		// and load them in revers order.
		block.sync();
		const int progress = i * BLOCK_SIZE + block.thread_rank();
		if (range.x + progress < range.y)
		{
			const int coll_id = point_list[range.y - progress - 1];
			collected_id[block.thread_rank()] = coll_id;
			collected_xy[block.thread_rank()] = points_xy_image[coll_id];
			collected_conic_opacity[block.thread_rank()] = conic_opacity[coll_id];
			collected_mu[block.thread_rank()] = mus[coll_id];
		}
		block.sync();

		// Iterate over Gaussians
		for (int j = 0; !done && j < min(BLOCK_SIZE, toDo); j++)
		{
			// Keep track of current Gaussian ID. Skip, if this one
			// is behind the last contributor for this pixel.
			contributor--;
			if (contributor >= last_contributor)
				continue;

			// Compute blending values, as before.
			const float2 xy = collected_xy[j];
			const float2 d = { xy.x - pixf.x, xy.y - pixf.y };
			const float4 con_o = collected_conic_opacity[j];
			float mu = collected_mu[j];
			const float power = -0.5f * (con_o.x * d.x * d.x + con_o.z * d.y * d.y) - con_o.y * d.x * d.y;
			if (power > 0.0f)
				continue;

			const float G = exp(power);

			// const float alpha = min(1.0f, con_o.w * mu * G);
			const float alpha = con_o.w * mu * G;
			if (alpha <0.00001f)
				continue;

			// Propagate gradients to per-Gaussian colors and keep
			// gradients w.r.t. alpha (blending factor for a Gaussian/pixel
			// pair).
			// Since we are simple sum, dchannel_dalpha = 1.0
			float dL_dalpha = 0.0f;
			const int global_id = collected_id[j];
			for (int ch = 0; ch < C; ch++)
			{
				const float dL_dchannel = dL_dpixel[ch];
				dL_dalpha += 1.f * dL_dchannel;
			}

			// Helpful reusable temporary variables
			const float dL_dG = con_o.w * mu * dL_dalpha;
			const float gdx = G * d.x;
			const float gdy = G * d.y;
			const float dG_ddelx = -gdx * con_o.x - gdy * con_o.y;
			const float dG_ddely = -gdy * con_o.z - gdx * con_o.y;

			// Update gradients w.r.t. 2D mean position of the Gaussian
			atomicAdd(&dL_dmean2D[global_id].x, dL_dG * dG_ddelx * ddelx_dx); // ddelx_dx is used to compensate ndc2pix
			atomicAdd(&dL_dmean2D[global_id].y, dL_dG * dG_ddely * ddely_dy); // ddelx_dx is used to compensate ndc2pix

			// Update gradients w.r.t. 2D covariance (2x2 matrix, symmetric)
			atomicAdd(&dL_dconic2D[global_id].x, -0.5f * gdx * d.x * dL_dG);
			atomicAdd(&dL_dconic2D[global_id].y, -1.0f * gdx * d.y * dL_dG); // gs code is 0.5f, which is futher compensated in computecov2D
			atomicAdd(&dL_dconic2D[global_id].w, -0.5f * gdy * d.y * dL_dG);

			// Update gradients w.r.t. opacity of the Gaussian
			atomicAdd(&(dL_dopacity[global_id]), mu * G * dL_dalpha);
			atomicAdd(&(dL_dmu[global_id]), con_o.w * G * dL_dalpha);
		}
	}
}


void BACKWARD::preprocess(
	int P,
	const float3* means3D,
	const int* radii,
	const glm::vec3* scales,
	const glm::vec4* rotations,
	const float scale_modifier,
	const float* cov3Ds,
	const float* viewmatrix,
	const float* projmatrix,
	const float focal_x, float focal_y,
	const float tan_fovx, float tan_fovy,
	const glm::vec3* campos,
	const float3* dL_dmean2D,
	const float* dL_dconic,
	const float* dL_dmu,
	glm::vec3* dL_dmean3D,
	float* dL_dcov3D,
	glm::vec3* dL_dscale,
	glm::vec4* dL_drot,
	const int mode)
{
	// Propagate gradients for the path of 2D conic matrix computation. 
	// Somewhat long, thus it is its own kernel rather than being part of 
	// "preprocess". When done, loss gradient w.r.t. 3D means has been
	// modified and gradient w.r.t. 3D covariance matrix has been computed.	
	computeCov2DCUDA << <(P + 255) / 256, 256 >> > (
		P,
		means3D,
		radii,
		cov3Ds,
		focal_x,
		focal_y,
		tan_fovx,
		tan_fovy,
		viewmatrix,
		dL_dconic,
		dL_dmu,
		(float3*)dL_dmean3D,
		dL_dcov3D,
		mode);

	// Propagate gradients for remaining steps: finish 3D mean gradients,
	// propagate color gradients to SH (if desireD), propagate 3D covariance
	// matrix gradients to scale and rotation.
	preprocessCUDA<NUM_CHANNELS> << < (P + 255) / 256, 256 >> > (
		P,
		(float3*)means3D,
		radii,
		(glm::vec3*)scales,
		(glm::vec4*)rotations,
		scale_modifier,
		projmatrix,
		campos,
		(float3*)dL_dmean2D,
		(glm::vec3*)dL_dmean3D,
		dL_dcov3D,
		dL_dscale,
		dL_drot);
}

void BACKWARD::render(
	const dim3 grid, const dim3 block,
	const uint2* ranges,
	const uint32_t* point_list,
	int W, int H,
	const float2* means2D,
	const float4* conic_opacity,
	const float* mus,
	const uint32_t* n_contrib,
	const float* dL_dpixels,
	float3* dL_dmean2D,
	float4* dL_dconic2D,
	float* dL_dopacity,
	float* dL_dmu
	)
{
	renderCUDA<NUM_CHANNELS> << <grid, block >> >(
		ranges,
		point_list,
		W, H,
		means2D,
		conic_opacity,
		mus,
		n_contrib,
		dL_dpixels,
		dL_dmean2D,
		dL_dconic2D,
		dL_dopacity,
		dL_dmu
		);
}

